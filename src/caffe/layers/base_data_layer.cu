#include <vector>
#include <opencv/cv.hpp>
#include "caffe/layers/base_data_layer.hpp"

namespace caffe {

template <typename Dtype>
void BasePrefetchingDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  Batch<Dtype>* batch = prefetch_full_.pop("Data layer prefetch queue empty");
  // Reshape to loaded data.
  top[0]->ReshapeLike(batch->data_);
  // Copy the data
  caffe_copy(batch->data_.count(), batch->data_.gpu_data(),
      top[0]->mutable_gpu_data());
  if (this->output_labels_) {
    // Reshape to loaded labels.
    top[1]->ReshapeLike(batch->label_);
    // Copy the labels.
    caffe_copy(batch->label_.count(), batch->label_.gpu_data(),
        top[1]->mutable_gpu_data());
  }
  // Ensure the copy is synchronous wrt the host, so that the next batch isn't
  // copied in meanwhile.
  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
/*
  Dtype *vis_data_0 = top[0]->mutable_cpu_data();

  vector<cv::Mat > vis_channels_0;
  int crop_size = 256;
  int stride = 8;

  for (int i = 0; i < 4; i++)
  {
	  cv::Mat vis_transformed_data(crop_size, crop_size, CV_32FC1, vis_data_0);
	  cv::Mat tmp;
	  vis_transformed_data.copyTo(tmp);
	  tmp = tmp * 256 + 128.0;
	  tmp.convertTo(tmp, CV_8U);
	  vis_channels_0.push_back(tmp);
	  vis_data_0 += crop_size * crop_size;
  }
  



  Dtype *vis_data = top[1]->mutable_cpu_data();
  vector<cv::Mat > vis_channels;
  for (int i = 0; i < 62; i++)
  {
	  cv::Mat vis_transformed_data(crop_size / stride, crop_size / stride, CV_32FC1, vis_data);

	  vis_channels.push_back(vis_transformed_data);
	  vis_data += crop_size * crop_size / (stride* stride);
	  cv::Mat tmp;
	  vis_transformed_data.copyTo(tmp);
	  tmp = tmp * 256;

	  tmp.convertTo(tmp, CV_8U);
	  char zz[256];
	  sprintf_s(zz, "F:/CoreLib/caffe-windows/Build/x64/Debug/%04d.jpg", i);
	  std::string str(zz);
	  imwrite(str, tmp);
  }


*/

  Dtype *vis_data_0 = top[0]->mutable_cpu_data();

  vector<cv::Mat > vis_channels_0;
  int crop_size = 120;
  int stride = 8;

  for (int i = 0; i < 3; i++)
  {
	  cv::Mat vis_transformed_data(112, 96, CV_32FC1, vis_data_0);
	  cv::Mat tmp;
	  vis_transformed_data.copyTo(tmp);
	 
	 //tmp.convertTo(tmp, CV_8U);
	  vis_channels_0.push_back(tmp);
	  vis_data_0 += 112 * 96;
  }


  Dtype *vis_data_1 = top[1]->mutable_cpu_data();

  cv::Mat show_img(112, 96, CV_32FC3);
  cv::merge(vis_channels_0, show_img);


  prefetch_free_.push(batch);
}

INSTANTIATE_LAYER_GPU_FORWARD(BasePrefetchingDataLayer);

}  // namespace caffe
